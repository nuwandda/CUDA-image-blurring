#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "wb.h"

// BLUR_SIZE is 1 because we span three rows, ( Row-1, Row, Row+1 )
#define BLUR_SIZE 1
#define BLOCK_SIZE 16

int xDimension;
int yDimension;

void writeImage(uchar4 *image, char *filename, char *memorytype);

void readImage(char *filename, uchar4 *image);


__global__ void unsharedBlurring(uchar4 *image, uchar4 *imageOutput, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over rows
    for (int x = 0; x < width; x++) {

        uchar4 pixel = make_uchar4(0, 0, 0, 0);
        float4 average = make_float4(0, 0, 0, 0);

        for (int i = -BLUR_SIZE; i <= BLUR_SIZE; i++) {
            for (int j = -BLUR_SIZE; j <= BLUR_SIZE; j++) {
                int blurRow = x + i;
                int blurCol = col + j;

                // Below, we check the boundary conditions
                if ((blurRow > -1) && (blurRow < width) && (blurCol > -1) && (blurCol < height)) {
                    pixel.x = image[blurRow + blurCol * height].x;
                    pixel.y = image[blurRow + blurCol * height].y;
                    pixel.z = image[blurRow + blurCol * height].z;
                } 
				else {
                    pixel = make_uchar4(0, 0, 0, 0);
                }
                average.x += pixel.x;
		        average.y += pixel.y;
		        average.z += pixel.z;
            }
        }
        // Divide summation to number of pixels
        average.x /= (float) (((BLUR_SIZE*2)+1)*((BLUR_SIZE*2)+1));
        average.y /= (float) (((BLUR_SIZE*2)+1)*((BLUR_SIZE*2)+1));
        average.z /= (float) (((BLUR_SIZE*2)+1)*((BLUR_SIZE*2)+1));

        imageOutput[x + col * height].x = (unsigned char) average.x;
        imageOutput[x + col * height].y = (unsigned char) average.y;
        imageOutput[x + col * height].z = (unsigned char) average.z;
        imageOutput[x + col * height].w = 255;
    }
}

__global__ void sharedBlurring(uchar4 *image, uchar4 *imageOutput, int width, int height) {
    int col = threadIdx.x + blockIdx.x * (blockDim.x - 2 * BLUR_SIZE);
    int row = threadIdx.y + blockIdx.y * (blockDim.y - 2 * BLUR_SIZE);
    uchar4 pixel = make_uchar4(0, 0, 0, 0);
    float4 average = make_float4(0, 0, 0, 0);

	if((row < height + BLUR_SIZE) && (col < width + BLUR_SIZE)) {
		// Allocate shared memory
		__shared__ uchar4 chunk[BLOCK_SIZE + (2 * BLUR_SIZE)][BLOCK_SIZE + (2 * BLUR_SIZE)];

		// Load elements into memory
		int relativeRow = row - BLUR_SIZE;
        int relativeCol = col - BLUR_SIZE;
        if ((relativeRow < height) && (relativeCol < width) && (relativeRow >= 0) && (relativeCol >= 0)) {
            chunk[threadIdx.y][threadIdx.x] = image[relativeRow*width + relativeCol];
        }
        else {
            chunk[threadIdx.y][threadIdx.x] = make_uchar4(0, 0, 0, 0);
        }

		__syncthreads();

		// Filter out-of-bounds threads
		if ((threadIdx.x >= BLUR_SIZE) && (threadIdx.y >= BLUR_SIZE) && (threadIdx.y < blockDim.y - BLUR_SIZE) && (threadIdx.x < blockDim.x - BLUR_SIZE)) {
			
			for (int i = -BLUR_SIZE; i <= BLUR_SIZE; i++) {
				for (int j = -BLUR_SIZE; j <= BLUR_SIZE; j++) {
					int blurRow = threadIdx.y + i;
					int blurCol = threadIdx.x + j;

					// Below, we check the boundary conditions
					if ((blurRow >= -1) && (blurRow < height) && (blurCol >= -1) && (blurCol < width)) {
                        pixel.x = chunk[blurRow][blurCol].x;
                        pixel.y = chunk[blurRow][blurCol].y;
                        pixel.z = chunk[blurRow][blurCol].z;
					}
                    else {
                        pixel = make_uchar4(0, 0, 0, 0);
                    }

                    average.x += pixel.x;
			        average.y += pixel.y;
			        average.z += pixel.z;
				}
			}
			// Divide summation to number of pixels
			average.x /= (float) (((BLUR_SIZE*2)+1)*((BLUR_SIZE*2)+1));
			average.y /= (float) (((BLUR_SIZE*2)+1)*((BLUR_SIZE*2)+1));
			average.z /= (float) (((BLUR_SIZE*2)+1)*((BLUR_SIZE*2)+1));

			imageOutput[relativeRow*width + relativeCol].x = (unsigned char) average.x;
			imageOutput[relativeRow*width + relativeCol].y = (unsigned char) average.y;
			imageOutput[relativeRow*width + relativeCol].z = (unsigned char) average.z;
			imageOutput[relativeRow*width + relativeCol].w = 255;
		}
	}
}

/**
 * Host main routine
 */
int main(int argc, char **argv) {
	if(argc != 3) {
		printf("Usage error. Program expects two arguments. \n");
    	printf("Usage: ./imageBlur IMAGENAME BLURTYPE(0 for unshared memory, 1 for shared memory) \n");
		printf("Usage Example: ./imageBlur 1.ppm 0 \n");
    	exit(1);
	}

    // System specifications
    printf("-->\n");
    printf("System Specifications:\n");
    printf("\tAzure NC6\n");
    printf("\tCores: 6\n");
    printf("\tGPU: Tesla K80\n");
    printf("\tMemory: 56 GB\n");
    printf("\tDisk: 380 GB SSD\n");
    printf("-->\n");
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    char *inputImageFile;
    wbImage_t inputImage;

    inputImageFile = argv[1];
    printf("Loading %s...\n", inputImageFile);
    inputImage = wbImport(inputImageFile);

    xDimension = wbImage_getWidth(inputImage);
    yDimension = wbImage_getHeight(inputImage);

    unsigned int imageSize, i;
    uchar4 *deviceImage, *deviceImageOutput, *deviceImageTemp;
    uchar4 *hostImage;
    hipEvent_t start, stop;
    float3 ms;

    imageSize = xDimension * yDimension * sizeof(uchar4);

    // Create event timers
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate and load host image
    hostImage = (uchar4 *) malloc(imageSize);
    // Verify that allocations succeeded
    if (hostImage == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    readImage(argv[1], hostImage);

    // Allocate device images
    err = hipMalloc((void **) &deviceImage, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **) &deviceImageOutput, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device image output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(start, 0);
	if (std::string(argv[2]) == "0") {
        printf("Executing blurring with unshared memory...\n");

		// Copy image to device memory
        printf("Copying image data from the host memory to the CUDA device...\n");
    	err = hipMemcpy(deviceImage, hostImage, imageSize, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy image from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        dim3 blocksPerGrid(xDimension / BLOCK_SIZE, 1);
    	dim3 threadsPerBlock(BLOCK_SIZE, 1);
        printf("CUDA kernel launching with {%d, %d} blocks of {%d, %d} threads...\n", blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x, threadsPerBlock.y);

		for (i = 0; i < 100; i++) {
			unsharedBlurring << <blocksPerGrid, threadsPerBlock >> >(deviceImage, deviceImageOutput, xDimension, yDimension);
			err = hipGetLastError();
            if (err != hipSuccess) {
                fprintf(stderr, "Failed to launch unsharedBlurring kernel (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }
			deviceImageTemp = deviceImage;
			deviceImage = deviceImageOutput;
			deviceImageOutput = deviceImageTemp;
		}
	}
	else {
        printf("Executing blurring with shared memory...\n");

		// Copy image to device memory
        printf("Copying image data from the host memory to the CUDA device...\n");
		err = hipMemcpy(deviceImage, hostImage, imageSize, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy image from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        dim3 blocksPerGrid(ceil(xDimension/(float)BLOCK_SIZE), ceil(yDimension/(float)BLOCK_SIZE), 1);
    	dim3 threadsPerBlock(BLOCK_SIZE + 2 * BLUR_SIZE, BLOCK_SIZE + 2 * BLUR_SIZE, 1);
        printf("CUDA kernel launching with {%d, %d, %d} blocks of {%d, %d, %d} threads...\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
        
        for (i = 0; i < 100; i++) {
            sharedBlurring << < blocksPerGrid, threadsPerBlock >> > (deviceImage, deviceImageOutput, xDimension, yDimension);
		    err = hipGetLastError();
            if (err != hipSuccess) {
                fprintf(stderr, "Failed to launch sharedBlurring kernel (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }
		    deviceImageTemp = deviceImage;
		    deviceImage = deviceImageOutput;
		    deviceImageOutput = deviceImageTemp;
        }
	}

    // Copy results back to host
    err = hipMemcpy(hostImage, deviceImage, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy images back to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms.x, start, stop);

    //output timings
    printf("Execution time:\n");
    if (std::string(argv[2]) == "0") {
        printf("\tUnshared version: %f\n", ms.x);
    }
    else {
        printf("\tShared version: %f\n", ms.x);
    }

    // Write image
    writeImage(hostImage, argv[1], argv[2]);

    // Free device memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    err = hipFree(deviceImage);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(deviceImageOutput);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device image output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free host memory
    free(hostImage);
    printf("DONE\n");

    return 0;
}

void writeImage(uchar4 *image, char *filename, char *memorytype) {
    FILE *f;

    char* str1;
    if (std::string(memorytype) == "0") {
        str1 = "unshared_output_";
    }
    else {
        str1 = "shared_output_";
    }
    char * str3 = (char *) malloc(1 + strlen(str1)+ strlen(filename) );
    strcpy(str3, str1);
    strcat(str3, filename);
    f = fopen(str3, "wb");
    if (f == NULL) {
        fprintf(stderr, "Error opening 'output.ppm' output file\n");
        exit(1);
    }
    fprintf(f, "P6\n");
    fprintf(f, "%d %d\n%d\n", xDimension, yDimension, 255);
    for (int x = 0; x < xDimension; x++) {
        for (int y = 0; y < yDimension; y++) {
            int i = x + y * yDimension;
            fwrite(&image[i], sizeof(unsigned char), 3, f);
        }
    }
    free(str3);
    fclose(f);
}

void readImage(char *filename, uchar4 *image) {
    FILE *f;
    char temp[256];
    unsigned int w, h, s;

    f = fopen(filename, "rb");
    if (f == NULL) {
        fprintf(stderr, "Error opening input file\n");
        exit(1);
    }
    printf("------\n");
    printf("Image Info:\n");
    fscanf(f, "%s\n", &temp);
    fscanf(f, "%*[^\n]\n");
    printf("\tType: %s\n", &temp);
    fscanf(f, "%d %d\n", &w, &h);
    printf("\tImage Size:%dx%d\n", w, h);
    fscanf(f, "%d\n", &s);
    printf("------\n");

    for (int x = 0; x < xDimension; x++) {
        for (int y = 0; y < yDimension; y++) {
            int i = x + y * yDimension;
            fread(&image[i], sizeof(unsigned char), 3, f);
        }
    }

    fclose(f);
}
